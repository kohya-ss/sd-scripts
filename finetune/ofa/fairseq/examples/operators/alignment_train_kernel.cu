#include "hip/hip_runtime.h"
/**
 * Copyright 2017-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h> // @manual=//caffe2/aten:ATen-cu
#include <hip/hip_runtime.h>
#include <algorithm> // std::min/max
#include <hipcub/hipcub.hpp>

#include "alignment_train_cuda.h"
#include "utils.h"

namespace {

// The thread block length in threads along the X dimension
constexpr int BLOCK_DIM_X = 128;
// The thread block length in threads along the Y dimension
constexpr int BLOCK_DIM_Y = 8;
// The thread block length in threads for scan operation
constexpr int SCAN_BLOCK = 512;

#define gpuErrchk(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }

inline void
gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
  if (code != hipSuccess) {
    fprintf(
        stderr,
        "\nGPUassert: %s %s %d\n",
        hipGetErrorString(code),
        file,
        line);
    if (abort)
      exit(code);
  }
}

template <typename T>
struct Prod {
  /// prod operator, returns <tt>a * b</tt>
  __host__ __device__ __forceinline__ T
  operator()(const T& a, const T& b) const {
    return a * b;
  }
};

template <typename T>
struct BlockPrefixProdCallbackOp {
  // Running prefix
  T running_total;

  // Constructor
  __device__ BlockPrefixProdCallbackOp(T running_total)
      : running_total(running_total) {}

  // Callback operator to be entered by the first warp of threads in the block.
  // Thread-0 is responsible for returning a value for seeding the block-wide
  // scan.
  __device__ T operator()(const T block_aggregate) {
    T old_prefix = running_total;
    running_total *= block_aggregate;
    return old_prefix;
  }
};

template <typename T>
struct BlockPrefixSumCallbackOp {
  // Running prefix
  T running_total;

  // Constructor
  __device__ BlockPrefixSumCallbackOp(T running_total)
      : running_total(running_total) {}

  // Callback operator to be entered by the first warp of threads in the block.
  // Thread-0 is responsible for returning a value for seeding the block-wide
  // scan.
  __device__ T operator()(const T block_aggregate) {
    T old_prefix = running_total;
    running_total += block_aggregate;
    return old_prefix;
  }
};

template <typename T>
__global__ void oneMinusPKernel(
    const T* __restrict__ p_choose,
    T* __restrict__ cumprod_1mp,
    uint32_t bsz,
    uint32_t tgt_len,
    uint32_t src_len) {
  for (uint32_t b = blockIdx.x; b < bsz; b += gridDim.x) {
    for (uint32_t tgt = threadIdx.y; tgt < tgt_len; tgt += blockDim.y) {
      for (uint32_t src = threadIdx.x; src < src_len; src += blockDim.x) {
        uint32_t idx = b * tgt_len * src_len + tgt * src_len + src;
        cumprod_1mp[idx] = 1 - p_choose[idx];
      }
    }
  }
}

template <typename T, int TPB>
__global__ void innermostScanKernel(
    T* __restrict__ cumprod_1mp,
    uint32_t bsz,
    uint32_t tgt_len,
    uint32_t src_len) {
  for (uint32_t b = blockIdx.y; b < bsz; b += gridDim.y) {
    for (uint32_t tgt = blockIdx.x; tgt < tgt_len; tgt += gridDim.x) {
      // Specialize BlockScan for a 1D block of TPB threads on type T
      typedef hipcub::BlockScan<T, TPB> BlockScan;
      // Allocate shared memory for BlockScan
      __shared__ typename BlockScan::TempStorage temp_storage;
      // Initialize running total
      BlockPrefixProdCallbackOp<T> prefix_op(1);

      const uint32_t tid = threadIdx.x;
      for (uint32_t block_src = 0; block_src < src_len;
           block_src += blockDim.x) {
        uint32_t src = block_src + tid;
        uint32_t idx = b * tgt_len * src_len + tgt * src_len + src;
        T thread_data = (src < src_len) ? cumprod_1mp[idx] : (T)0;

        // Collectively compute the block-wide inclusive prefix sum
        BlockScan(temp_storage)
            .ExclusiveScan(thread_data, thread_data, Prod<T>(), prefix_op);
        __syncthreads();

        // write the scanned value to output
        if (src < src_len) {
          cumprod_1mp[idx] = thread_data;
        }
      }
    }
  }
}

template <typename T>
__global__ void clampKernel(
    const T* __restrict__ cumprod_1mp,
    T* __restrict__ cumprod_1mp_clamp,
    uint32_t bsz,
    uint32_t tgt_len,
    uint32_t src_len,
    T min_val,
    T max_val) {
  for (uint32_t b = blockIdx.x; b < bsz; b += gridDim.x) {
    for (uint32_t tgt = threadIdx.y; tgt < tgt_len; tgt += blockDim.y) {
      for (uint32_t src = threadIdx.x; src < src_len; src += blockDim.x) {
        uint32_t idx = b * tgt_len * src_len + tgt * src_len + src;
        if (cumprod_1mp[idx] < min_val) {
          cumprod_1mp_clamp[idx] = min_val;
        } else if (cumprod_1mp[idx] > max_val) {
          cumprod_1mp_clamp[idx] = max_val;
        } else {
          cumprod_1mp_clamp[idx] = cumprod_1mp[idx];
        }
      }
    }
  }
}

template <typename T>
__global__ void initAlphaCUDAKernel(
    T* alpha,
    uint32_t bsz,
    uint32_t tgt_len,
    uint32_t src_len) {
  // alpha[:, 0, 0] = 1.0
  for (uint32_t b = blockIdx.x; b < bsz; b += gridDim.x) {
    alpha[b * tgt_len * src_len] = (T)1.0;
  }
}

template <typename T, int TPB>
__global__ void alignmentTrainCUDAKernel(
    const T* __restrict__ p_choose,
    const T* __restrict__ cumprod_1mp,
    const T* __restrict__ cumprod_1mp_clamp,
    T* __restrict__ alpha,
    uint32_t bsz,
    uint32_t tgt_len,
    uint32_t src_len,
    uint32_t tgt) {
  for (uint32_t b = blockIdx.x; b < bsz; b += gridDim.x) {
    // Specialize BlockScan for a 1D block of TPB threads on type T
    typedef hipcub::BlockScan<T, TPB> BlockScan;

    // Allocate shared memory for BlockScan
    __shared__ typename BlockScan::TempStorage temp_storage;
    // Initialize running total
    BlockPrefixSumCallbackOp<T> prefix_op(0);

    uint32_t b_offset = b * tgt_len * src_len;
    const uint32_t tid = threadIdx.x;
    for (uint32_t block_src = 0; block_src < src_len; block_src += blockDim.x) {
      uint32_t src = block_src + tid;
      // Obtain a segment of consecutive items that are blocked across threads
      uint32_t inout_idx, alpha_idx;
      if (tgt == 0) {
        // both alpha and other input index is [b][0][src]
        alpha_idx = b_offset + src;
      } else {
        // alpha index is [b][tgt-1][src]
        alpha_idx = b_offset + (tgt - 1) * src_len + src;
      }
      inout_idx = b_offset + tgt * src_len + src;
      T thread_data = (T)0;
      if (src < src_len) {
        thread_data = alpha[alpha_idx] / cumprod_1mp_clamp[inout_idx];
      }

      // Collectively compute the block-wide inclusive prefix sum
      BlockScan(temp_storage).InclusiveSum(thread_data, thread_data, prefix_op);
      __syncthreads();

      if (src < src_len) {
        T out = thread_data * p_choose[inout_idx] * cumprod_1mp[inout_idx];
        // Clamps all elements into the range [ 0, 1.0 ]
        alpha[inout_idx] = std::min<T>(std::max<T>(out, 0), (T)1.0);
      }
    }
  }
}

template <typename T>
void exclusiveCumprod(
    const T* p_choose,
    T* cumprod_1mp,
    uint32_t bsz,
    uint32_t tgt_len,
    uint32_t src_len,
    uint32_t max_grid_x,
    uint32_t max_grid_y,
    hipStream_t& stream) {
  // cumprod_1mp = 1 - p_choose
  dim3 grid(std::min<T>(max_grid_x, bsz), 1, 1);
  dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
  oneMinusPKernel<T><<<grid, block, 0, stream>>>(
      p_choose, cumprod_1mp, bsz, tgt_len, src_len);
  gpuErrchk(hipGetLastError());

  // scan on the innermost dimension of cumprod_1mp
  // cumprod_1mp = cumprod(cumprod_1mp)
  dim3 grid_scan(
      std::min<T>(max_grid_x, tgt_len), std::min<T>(max_grid_y, bsz), 1);
  innermostScanKernel<T, SCAN_BLOCK><<<grid_scan, SCAN_BLOCK, 0, stream>>>(
      cumprod_1mp, bsz, tgt_len, src_len);
  gpuErrchk(hipGetLastError());
}

template <typename T>
void alignmentTrainCUDAImpl(
    const T* p_choose,
    T* alpha,
    uint32_t bsz,
    uint32_t tgt_len,
    uint32_t src_len,
    float eps) {
  // p_choose: bsz , tgt_len, src_len
  // cumprod_1mp: bsz , tgt_len, src_len
  // cumprod_1mp_clamp : bsz, tgt_len, src_len
  // alpha: bsz, tgt_len, src_len
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  uint32_t max_grid_x = at::cuda::getCurrentDeviceProperties()->maxGridSize[0];
  uint32_t max_grid_y = at::cuda::getCurrentDeviceProperties()->maxGridSize[1];

  // Implementing exclusive cumprod.
  // cumprod_1mp = cumprod(1 - p_choose)
  // There is cumprod in pytorch, however there is no exclusive mode.
  // cumprod(x) = [x1, x1x2, x2x3x4, ..., prod_{i=1}^n x_i]
  // exclusive means
  // cumprod(x) = [1, x1, x1x2, x1x2x3, ..., prod_{i=1}^{n-1} x_i]
  uint32_t elements = bsz * tgt_len * src_len;
  T* cumprod_1mp;
  gpuErrchk(hipMalloc(&cumprod_1mp, elements * sizeof(T)));
  exclusiveCumprod<T>(
      p_choose,
      cumprod_1mp,
      bsz,
      tgt_len,
      src_len,
      max_grid_x,
      max_grid_y,
      stream);

  // clamp cumprod_1mp to the range [eps, 1.0]
  T* cumprod_1mp_clamp;
  gpuErrchk(hipMalloc(&cumprod_1mp_clamp, elements * sizeof(T)));
  dim3 grid_clamp(std::min<T>(max_grid_x, bsz), 1, 1);
  dim3 block_clamp(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
  clampKernel<T><<<grid_clamp, block_clamp, 0, stream>>>(
      cumprod_1mp, cumprod_1mp_clamp, bsz, tgt_len, src_len, (T)eps, (T)1.0);
  gpuErrchk(hipGetLastError());

  // ai = p_i * cumprod(1 − pi) * cumsum(a_i / cumprod(1 − pi))
  dim3 grid_init(std::min<int>(max_grid_x, bsz), 1, 1);
  initAlphaCUDAKernel<T>
      <<<grid_init, 1, 0, stream>>>(alpha, bsz, tgt_len, src_len);
  gpuErrchk(hipGetLastError());

  const int grid = std::min(bsz, max_grid_x);

  for (uint32_t i = 0; i < tgt_len; i++) {
    alignmentTrainCUDAKernel<T, SCAN_BLOCK><<<grid, SCAN_BLOCK, 0, stream>>>(
        p_choose,
        cumprod_1mp,
        cumprod_1mp_clamp,
        alpha,
        bsz,
        tgt_len,
        src_len,
        i);
    gpuErrchk(hipGetLastError());
  }

  gpuErrchk(hipFree(cumprod_1mp));
  gpuErrchk(hipFree(cumprod_1mp_clamp));
}

} // namespace

void alignmentTrainCUDAWrapper(
    const torch::Tensor& p_choose,
    torch::Tensor& alpha,
    float eps) {
  // p_choose dimension: bsz, tgt_len, src_len
  uint32_t bsz = p_choose.size(0);
  uint32_t tgt_len = p_choose.size(1);
  uint32_t src_len = p_choose.size(2);

  hipSetDevice(p_choose.get_device());

  AT_DISPATCH_FLOATING_TYPES_AND2(
      torch::ScalarType::Half,
      torch::ScalarType::BFloat16,
      p_choose.scalar_type(),
      "alignmentTrainCUDAImpl",
      [&]() {
        alignmentTrainCUDAImpl<scalar_t>(
            p_choose.data_ptr<scalar_t>(),
            alpha.data_ptr<scalar_t>(),
            bsz,
            tgt_len,
            src_len,
            eps);
      });
}
