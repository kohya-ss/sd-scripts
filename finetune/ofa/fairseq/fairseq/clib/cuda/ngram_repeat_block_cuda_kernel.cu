#include "hip/hip_runtime.h"
/*
Copyright (c) Microsoft Corporation.
Licensed under the MIT License.
*/

/*
Kernel implementation for blocking repeated n-grams.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <torch/extension.h>
#include <vector>

// Ban repeated ngrams of length = 'no_repeat_ngram_size'
__global__ void banRepeatedTokens(
    long* __restrict__ tokens,
    float* __restrict__ lprobs,
    int max_predict_len,
    int vocab_size,
    int no_repeat_ngram_size) {
  auto row = blockIdx.x;
  auto col = threadIdx.x;
  auto start = row * (max_predict_len) + col;
  // Each thread compares ngram starting from
  // thread index with final ngram starting from
  // step - no_repeat_ngram_size +2
  auto check_start_pos = blockDim.x;
  auto lprob_start = row * vocab_size;
  bool is_banned = true;
  extern __shared__ long tokens_shm[];
  tokens_shm[col] = tokens[start];
  if (col == blockDim.x - 1) {
    for (int i = 1; i < no_repeat_ngram_size; i++) {
      if (col + i < max_predict_len) {
        tokens_shm[col + i] = tokens[start + i];
      }
    }
  }
  __syncthreads();

  for (int k = 0; k < no_repeat_ngram_size - 1; k++) {
    if (tokens_shm[col + k] != tokens_shm[check_start_pos + k]) {
      is_banned = false;
    }
  }
  if (is_banned == true) {
    auto token_to_be_banned = tokens_shm[col + no_repeat_ngram_size - 1];
    lprobs[lprob_start + token_to_be_banned] = -INFINITY;
  }
}

// Allocate blocks and threads based on
// batch size and sequence length and launch
// kernel
torch::Tensor ngram_repeat_block_cuda_forward(
    const torch::Tensor tokens,
    torch::Tensor lprobs,
    int bsz,
    int step,
    int beam_size,
    int no_repeat_ngram_size) {
  int threads = step - no_repeat_ngram_size + 2;
  if (threads <= 0)
    return lprobs;
  int max_predict_len = tokens.size(1);
  int vocab_size = lprobs.size(1);
  auto token_ptr = tokens.data_ptr<long>();
  auto lprob_ptr = lprobs.data_ptr<float>();
  int blocks = bsz * beam_size;
  int shared_mem_size = (step + 1) * sizeof(long);

  // Launching N blocks where N is number of samples in a batch (beams*bsz)
  // Launching T threads where T is number of previous ngrams in a sample
  // Allocating shared mem per block for fastser access of input tokens since
  // each token will be accessed N times to compare with current Ngram where
  // N is Ngram size.
  banRepeatedTokens<<<blocks, threads, shared_mem_size>>>(
      token_ptr, lprob_ptr, max_predict_len, vocab_size, no_repeat_ngram_size);
  return lprobs;
}
