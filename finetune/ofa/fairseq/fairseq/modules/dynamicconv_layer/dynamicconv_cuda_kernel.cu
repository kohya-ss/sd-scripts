#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "../cuda_utils.cu"
#include "dynamicconv_cuda.cuh"
#include "dynamicconv_cuda_backward.cu"
#include "dynamicconv_cuda_forward.cu"

// FS is filter size and kernels are specialized for filter sizes
template <int FS, int SB, int padding_l, typename scalar_t>
__global__ void dynamicconv_forward_kernel(
    const scalar_t* input,
    const scalar_t* weight,
    int minibatch,
    int sequenceLength,
    int numFeatures,
    int numFiltersInBlock,
    int numHeads,
    scalar_t* output) {
  assert(blockDim.x == SB);

  const int tid = threadIdx.x;
  const int batchIdx = blockIdx.x;
  const int featureIdx = blockIdx.y;
  const int head = featureIdx / numFiltersInBlock;

  const int IOOffset =
      batchIdx * numFeatures * sequenceLength + featureIdx * sequenceLength;
  const scalar_t* inputFeature = &input[IOOffset];
  scalar_t* outputFeature = &output[IOOffset];

  scalar_t filter[FS];

  __shared__ scalar_t tempInput[SB + FS];
  zeroSharedMem<FS, SB, padding_l>(tempInput);

  const int numIterations = divUp<int, int>(sequenceLength, SB);

  for (int i = 0; i < numIterations; ++i) {
    __syncthreads();
    const int inputOffset = i * SB;
    load_input_to_shared<FS, SB, padding_l>(
        inputFeature,
        inputOffset,
        sequenceLength,
        i,
        numIterations,
        false,
        tempInput);
    __syncthreads();
    if (inputOffset + tid < sequenceLength) {
#pragma unroll
      for (int k = 0; k < FS; ++k) {
        const int filterOffset = batchIdx * numHeads * FS * sequenceLength +
            head * FS * sequenceLength + k * sequenceLength + i * SB + tid;
        filter[k] = weight[filterOffset];
      }

      scalar_t out = scalar_t(0.0);
#pragma unroll
      for (int k = 0; k < FS; ++k) {
        out += filter[k] * tempInput[tid + k];
      }

      outputFeature[inputOffset + tid] = out;
    }
  }
}

template <int FS, int SB, int padding_l, typename scalar_t>
__global__ void dynamicconv_backward_kernel(
    const scalar_t* gradOutput, // B * C * T
    const scalar_t* input, // B * C * T
    const scalar_t* weight,
    int minibatch,
    int sequenceLength,
    int numFeatures,
    int numFiltersInBlock,
    int numHeads,
    scalar_t* gradWeight,
    scalar_t* gradInput) { // B * H * k * T

  assert(blockDim.x == SB);

  // each block operates on a single batch and filter head
  const int tid = threadIdx.x;
  const int batchIdx = blockIdx.x;
  const int headIdx = blockIdx.y;
  const int chunkIdx = blockIdx.z;

  const int numChunks = divUp<int, int>(sequenceLength, SB);
  const int inputOffset = chunkIdx * SB;

  // initialize shared memory for output gradient and input
  __shared__ scalar_t tempGradOutput[SB + FS];
  __shared__ scalar_t tempInput[SB + FS];
  const int padding = FS - padding_l - 1;

  zeroSharedMem<FS, SB, padding>(tempGradOutput);
  zeroSharedMem<FS, SB, padding_l>(tempInput);

  // initialize local filter and weight gradient sum arrays
  scalar_t tempGradSum[FS];
  scalar_t bfilter[FS];
  for (int k = 0; k < FS; ++k) {
    tempGradSum[k] = scalar_t(0.0);

    int idxOffset = inputOffset + tid + k - padding;
    if (idxOffset >= 0 && idxOffset < sequenceLength) {
      int bfilterOffset = batchIdx * numHeads * FS * sequenceLength +
          headIdx * FS * sequenceLength + (FS - k - 1) * sequenceLength +
          idxOffset;
      bfilter[k] = weight[bfilterOffset];
    } else {
      bfilter[k] = scalar_t(0.0);
    }
  }

  // iterate over filter block
  for (int featureIdx = 0; featureIdx < numFiltersInBlock; ++featureIdx) {
    __syncthreads();

    // load input and output gradient for this channel and chunk
    const int IOOffset = batchIdx * numFeatures * sequenceLength +
        (headIdx * numFiltersInBlock + featureIdx) * sequenceLength;
    const scalar_t* inputFeature = &input[IOOffset];
    const scalar_t* gradOutputFeature = &gradOutput[IOOffset];
    scalar_t* gradInputFeature = &gradInput[IOOffset];

    load_input_to_shared<FS, SB, padding>(
        gradOutputFeature,
        inputOffset,
        sequenceLength,
        chunkIdx,
        numChunks,
        true,
        tempGradOutput);
    load_input_to_shared<FS, SB, padding_l>(
        inputFeature,
        inputOffset,
        sequenceLength,
        chunkIdx,
        numChunks,
        true,
        tempInput);
    __syncthreads();

    // sum input and weight gradients
    scalar_t out = scalar_t(0.0);
#pragma unroll
    for (int k = 0; k < FS; ++k) {
      tempGradSum[k] += tempInput[tid + k] * tempGradOutput[tid + padding];
      out += bfilter[k] * tempGradOutput[tid + k];
    }

    if (inputOffset + tid < sequenceLength) {
      gradInputFeature[inputOffset + tid] = out;
    }
  }

  const int gradOffset =
      batchIdx * numHeads * FS * sequenceLength + headIdx * FS * sequenceLength;
  scalar_t* gradWeightFeature = &gradWeight[gradOffset];

  // write weight gradient
  if (inputOffset + tid < sequenceLength) {
    for (int k = 0; k < FS; ++k) {
      const int outputOffset = k * sequenceLength + inputOffset + tid;
      gradWeightFeature[outputOffset] = tempGradSum[k];
    }
  }
}
