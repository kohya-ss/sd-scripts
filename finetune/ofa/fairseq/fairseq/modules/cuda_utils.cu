#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

template <typename U, typename V>
constexpr __host__ __device__ auto divUp(U a, V b) -> decltype(a + b) {
  return (a + b - 1) / b;
}

template <int FS, int SB, int padding_l, typename scalar_t>
__inline__ __device__ void zeroSharedMem(scalar_t* data) {
  /*
    Given an array of length FS + SB, zero out the first padding_l and last
    (FS - padding_l) values in the array
  */

  int tid = threadIdx.x;

  if (FS < SB) {
    // zero all if we have enough threads in a block to do all of them
    if (tid < padding_l || tid > SB - FS + padding_l - 1) {
      data[tid] = scalar_t(0.0);
    }
  } else {
    // otherwise zero out one block at a time
    const int numIterations = divUp<int, int>(FS, SB);
    for (int i = 0; i < numIterations; i++) {
      int offset = i * SB;
      if (tid + offset < padding_l) {
        data[tid + offset] = scalar_t(0.0);
      } else if (tid + offset < FS) {
        data[SB + tid + offset] = scalar_t(0.0);
      }
    }
  }
}

template <typename scalar_t>
__inline__ __device__ scalar_t warpReduce(scalar_t data) {
  /*
    Reduce an array within each warp. After processing all values in warp will
    caontain the sum of all original values in that warp.

    data - pointer to data to reduce
  */
  data += __shfl_xor_sync(SHFL_MASK, data, 16);
  data += __shfl_xor_sync(SHFL_MASK, data, 8);
  data += __shfl_xor_sync(SHFL_MASK, data, 4);
  data += __shfl_xor_sync(SHFL_MASK, data, 2);
  data += __shfl_xor_sync(SHFL_MASK, data, 1);
  return data;
}

template <typename scalar_t>
__inline__ __device__ scalar_t blockReduce(scalar_t data) {
  /*
     Reduce an entire array on the block level. After processing, the
     first value in the array will contain the reduced sum.

     data - pointer to data to reduce
  */

  static __shared__ scalar_t warpSum[32];
  const int tid = threadIdx.x;
  int wid = tid / 32;
  int lane = tid % 32;

  __syncthreads();

  // reduce each warp then write to shared memory
  scalar_t sum = warpReduce(data);
  if (lane == 0) {
    warpSum[wid] = sum;
  }

  __syncthreads();

  scalar_t v;
  // perform final sum of partial warp sums
  if (tid < blockDim.x / 32) {
    v = warpSum[lane];
  } else {
    v = scalar_t(0.0);
  }

  if (wid == 0) {
    v = warpReduce(v);
  }
  __syncthreads();

  return v;
}

void checkCudaStatus(hipError_t status, int lineNumber = -1) {
  if (status != hipSuccess) {
    std::cout << hipGetErrorString(status) << " at line " << lineNumber
              << std::endl;
    std::cout << "Exiting" << std::endl;
    exit(1);
  }
}

template <int FS, int SB, int padding_l, typename scalar_t>
__device__ void load_input_to_shared(
    const scalar_t* input, // global memory
    int inputOffset,
    int sequenceLength,
    int iteration,
    int numIterations,
    bool no_prev,
    scalar_t* output /* shared memory */) {
  /*
    Load a block size of input into shared memory with
    right and left overhang of total size FS. If previously
    loaded memory, overlap will be shifted over to reduce
    global memory access

    input - pointer to start of channel sequence
    inputOffset - how far in the sequence to start loading
    sequenceLength - total length of sequence
    iteration - which block of sequence we are loading
    numIterations - total number of blocks to load
    no_prev - whether to load the whole block if the previous block
              wasn't loaded
    output - shared memory to write input to
  */

  const int tid = threadIdx.x;

  // Load the left "overhang" of input
  if (iteration > 0) {
    if (padding_l < SB) {
      // load all at once
      if (tid < padding_l) {
        output[tid] =
            (no_prev) ? input[inputOffset - padding_l + tid] : output[tid + SB];
      }
    } else {
      // load in chunks of size SB
      int numIterations = divUp<int, int>(padding_l, SB);
      for (int i = 0; i < numIterations; i++) {
        int offset = i * SB;
        if ((tid + offset) < padding_l) {
          output[tid + offset] = (no_prev)
              ? input[inputOffset - padding_l + tid + offset]
              : output[tid + offset + SB];
        }
      }
    }
  }

  // Load the right "overhang" of input
  if (iteration < (numIterations - 1)) {
    const int elementsLeft = sequenceLength - (iteration + 1) * SB;

    if ((FS - padding_l) < SB) {
      // load all at once
      if (tid < (FS - padding_l)) {
        output[padding_l + SB + tid] = (tid < elementsLeft)
            ? input[inputOffset + SB + tid]
            : scalar_t(0.0);
      }
    } else {
      // load in chunks of size SB
      int numIterations = divUp<int, int>(FS - padding_l, SB);
      for (int i = 0; i < numIterations; i++) {
        int offset = i * SB;
        if ((tid + offset) < (FS - padding_l)) {
          output[padding_l + SB + tid + offset] =
              ((tid + offset) < elementsLeft)
              ? input[inputOffset + SB + tid + offset]
              : scalar_t(0.0);
        }
      }
    }
  }

  // We should also clear out the right "overhang"
  if (iteration == (numIterations - 1)) {
    if ((FS - padding_l) < SB) {
      // clear out all at once
      if (tid < (FS - padding_l)) {
        output[padding_l + SB + tid] = scalar_t(0.0);
      }
    } else {
      // clear in chunks of size SB
      int numIterations = divUp<int, int>(FS - padding_l, SB);
      for (int i = 0; i < numIterations; i++) {
        int offset = i * SB;
        if ((tid + offset) < (FS - padding_l)) {
          output[padding_l + SB + tid + offset] = scalar_t(0.0);
        }
      }
    }
  }
  output[tid + padding_l] = ((inputOffset + tid) < sequenceLength)
      ? input[inputOffset + tid]
      : scalar_t(0.0);
}
